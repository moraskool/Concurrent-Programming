#include "hip/hip_runtime.h"
// cuda headers
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// created files
#include "cuda_common.cuh" 

// sum the arrays on the GPU
__global__ void inclusiveScan(int* input, int* output, int size)
{
    output[0] = input[0];
    for (int i = 0; i < size; i++)
    {
        output[i] = output[i - 1] + input[i];
    }


}

int main()
{}